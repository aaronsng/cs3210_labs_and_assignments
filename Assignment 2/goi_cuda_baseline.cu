#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <ctype.h>
#include <errno.h>
#include <time.h>
#include "util.h"
#include "exporter.h"
#include "settings.h"

// including the "dead faction": 0
#define MAX_FACTIONS 10

// this macro is here to make the code slightly more readable, not because it can be safely changed to
// any integer value; changing this to a non-zero value may break the code
#define DEAD_FACTION 0

/*
 * Gets the value at. GPU implementation
 */
__device__ int getValueAtDevice(const int * grid, int nRows, int nCols, int row, int col) 
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols) 
    {
        return -1; 
    }
    return *(grid + (row * nCols) + col);
}

/*
 * Sets the value at. GPU implementation
 */
__device__ void setValueAtDevice(int * grid, int nRows, int nCols, int row, int col, int val) 
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols) 
    {
        return; 
    }
    *(grid + (row * nCols) + col) = val;
}

/*
 * Specifies the number(s) of live neighbors of the same faction required for a dead cell to become alive.
 */
__device__ bool isBirthable(int n)
{
    return n == 3;
}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a live cell to remain alive.
 */
__device__ bool isSurvivable(int n)
{
    return n == 2 || n == 3;
}

/**
 * Specifies the number of live neighbors of a different faction required for a live cell to die due to fighting.
 */
__device__ bool willFight(int n) {
    return n > 0;
}

/**
 * Computes and returns the next state of the cell specified by row and col based on currWorld and invaders. Sets *diedDueToFighting to
 * true if this cell should count towards the death toll due to fighting.
 * currWorld is a variable in the global memory.
 * invaders can be NULL if there are no invaders.
 */
__device__ int getNextState(const int *currWorld, const int *invaders, int nRows, int nCols, int row, int col, bool *diedDueToFighting)
{
    // we'll explicitly set if it was death due to fighting
    *diedDueToFighting = false;

    // faction of this cell
    int cellFaction = getValueAtDevice(currWorld, nRows, nCols, row, col);

    // did someone just get landed on?
    if (invaders != NULL && getValueAtDevice(invaders, nRows, nCols, row, col) != DEAD_FACTION)
    {
        *diedDueToFighting = cellFaction != DEAD_FACTION;
        return getValueAtDevice(invaders, nRows, nCols, row, col);
    }

    // tracks count of each faction adjacent to this cell
    int neighborCounts[MAX_FACTIONS];
    memset(neighborCounts, 0, MAX_FACTIONS * sizeof(int));

    // count neighbors (and self)
    for (int dy = -1; dy <= 1; dy++)
    {
        for (int dx = -1; dx <= 1; dx++)
        {
            int faction = getValueAtDevice(currWorld, nRows, nCols, row + dy, col + dx);
            bool faction_is_dead = faction >= DEAD_FACTION;
            neighborCounts[faction] += faction_is_dead;
        }
    }

    // we counted this cell as its "neighbor"; adjust for this
    neighborCounts[cellFaction]--;

    if (cellFaction == DEAD_FACTION)
    {
        // this is a dead cell; we need to see if a birth is possible:
        // need exactly 3 of a single faction; we don't care about other factions

        // by default, no birth
        int newFaction = DEAD_FACTION;

        // start at 1 because we ignore dead neighbors
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            int count = neighborCounts[faction];
            if (isBirthable(count))
            {
                newFaction = faction;
            }
        }

        return newFaction;
    }
    else
    {
        /** 
         * this is a live cell; we follow the usual rules:
         * Death (fighting): > 0 hostile neighbor
         * Death (underpopulation): < 2 friendly neighbors and 0 hostile neighbors
         * Death (overpopulation): > 3 friendly neighbors and 0 hostile neighbors
         * Survival: 2 or 3 friendly neighbors and 0 hostile neighbors
         */

        int hostileCount = 0;
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            if (faction == cellFaction)
            {
                continue;
            }
            hostileCount += neighborCounts[faction];
        }

        if (willFight(hostileCount))
        {
            *diedDueToFighting = true;
            return DEAD_FACTION;
        }

        int friendlyCount = neighborCounts[cellFaction];
        if (!isSurvivable(friendlyCount))
        {
            return DEAD_FACTION;
        }

        return cellFaction;
    }
}

/** 
 * Main kernel code: 
 * Breakup the code to the element wise. The kernel will operate element wise operations. 
 * currentGeneration -> Located in global memory
 * invasionPlans -> Located in global memory
 * invasionTimes -> Located in global memory
 */
__global__ void simulate_per_thread(int * world, int * wholeNewWorld, int * invasionPlans, int * nRows, int * nCols, int * row_start_collection, int * row_end_collection, int * col_start_collection, int * col_end_collection, int * death_toll_collection) 
{
    // localised death toll, to minimise branching
    int deathToll_ = 0;

    // blockIndex will divide the problem as in Assignment 1. The general strategy in Assignment 1 were
    // coarsely grained tasks performed over the row
    int blockIndex = blockIdx.x * gridDim.y * gridDim.z + blockIdx.y * gridDim.z + blockIdx.z;
    int rowStart = row_start_collection[blockIndex];
    int rowEnd = row_end_collection[blockIndex];

    // threadIndex will divide the problem further. Now instead of performing over each individual row
    // GPU programming calls for greater granularity, and hence each individual row maybe too big a task
    // Hence, each individual GPU thread will reduce the task further by chopping up the problem with column wise distribution.
    int threadIndex = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;
    int colStart = col_start_collection[threadIndex];
    int colEnd = col_end_collection[threadIndex];

    //printf("\n%d %d\n", blockIndex, threadIndex);
    //printf("\n%d %d %d %d %d %d\n", rowStart, rowEnd, colStart, colEnd, blockIndex, threadIndex);

    // check if there is an invasion invoked in this kernel call
    for (int row = rowStart; row < rowEnd; row++) 
    {
        for (int col = colStart; col < colEnd; col++)
        {
            bool diedDueToFighting;
            int nextState = getNextState(world, invasionPlans, *nRows, *nCols, row, col, &diedDueToFighting);
            setValueAtDevice(wholeNewWorld, *nRows, *nCols, row, col, nextState);
            
            // This is possible as booleans have a value of 1. reduce branching within kernel as well
            deathToll_ += diedDueToFighting; 
        }
    }
    death_toll_collection[blockIndex * blockDim.x * blockDim.y * blockDim.z + threadIndex] += deathToll_;
}

/**
 * Check CUDA errors
 */
void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess) 
    {
        printf("Last CUDA error: %s\n", hipGetErrorString(rc));
    }
}

/**
 * The main simulation logic.
 * 
 * goi does not own startWorld, invasionTimes or invasionPlans and should not modify or attempt to free them.
 * nThreads is the number of threads to simulate with. It is ignored by the sequential implementation.
 */
int goi(int nThreads, int nGenerations, const int *startWorld, int nRows, int nCols, int nInvasions, const int *invasionTimes, int **invasionPlans, dim3 gridDim_, dim3 blockDim_)
{ 
    // calculate wall clock
    // struct timespec begin, end;
    // clock_gettime(CLOCK_REALTIME, &begin);

    // send nRows over
    int * d_n_rows;
    hipMalloc((void**) &d_n_rows, sizeof(int));
    hipMemcpy(d_n_rows, &nRows, sizeof(int), hipMemcpyHostToDevice);

    int * d_n_cols;
    hipMalloc((void**) &d_n_cols, sizeof(int));
    hipMemcpy(d_n_cols, &nCols, sizeof(int), hipMemcpyHostToDevice);

    // initialise the world on the device
    int * d_world;
    hipMalloc((void**) &d_world, nRows * nCols * sizeof(int));
    hipMemcpy(d_world, startWorld, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);

    // start computing the start and end indices of the various rows
    int * d_row_start_collection = NULL;
    int * d_row_end_collection = NULL;
    int blockCount = gridDim_.x * gridDim_.y * gridDim_.z;
    int * row_start_collection;
    int * row_end_collection;
    int baseRowWidth = nRows / blockCount;
    int remnant = nRows % blockCount;
    int startIndex, endIndex = 0;
    hipHostMalloc((void **) &row_start_collection, sizeof(int) * blockCount);
    hipHostMalloc((void **) &row_end_collection, sizeof(int) * blockCount);
    
    for (int i = 0; i < blockCount; i++) 
    {
        startIndex = endIndex;
        if (i < (blockCount - remnant)) 
        {
            endIndex = (startIndex + baseRowWidth) > nRows ? nRows : (startIndex + baseRowWidth);
        }
        else 
        {
            endIndex = (startIndex + baseRowWidth + 1) > nRows ? nRows : (startIndex + baseRowWidth + 1);
        }

        row_start_collection[i] = startIndex;
        row_end_collection[i] = endIndex;
    }

    // malloc and copy over
    hipMalloc((void**) &d_row_start_collection, blockCount * sizeof(int));
    hipMemcpy(d_row_start_collection, row_start_collection, blockCount * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_row_end_collection, blockCount * sizeof(int));
    hipMemcpy(d_row_end_collection, row_end_collection, blockCount * sizeof(int), hipMemcpyHostToDevice);
    
    // start computing the start and end indices of the various columns
    int * d_col_start_collection = NULL;
    int * d_col_end_collection = NULL;
    int threadCount = blockDim_.x * blockDim_.y * blockDim_.z;
    int * col_start_collection;
    int * col_end_collection;
    int baseColWidth = nCols / threadCount;
    remnant = nCols % threadCount;
    endIndex = 0;
    hipHostMalloc((void **) &col_start_collection, sizeof(int) * threadCount);
    hipHostMalloc((void **) &col_end_collection, sizeof(int) * threadCount);
    
    for (int i = 0; i < threadCount; i++) 
    {
        startIndex = endIndex;
        if (i < (threadCount - remnant)) 
        {
            endIndex = (startIndex + baseColWidth) > nCols ? nCols : (startIndex + baseColWidth);
        }
        else 
        {
            endIndex = (startIndex + baseColWidth + 1) > nCols ? nCols : (startIndex + baseColWidth + 1);
        }

        col_start_collection[i] = startIndex;
        col_end_collection[i] = endIndex;
    }

    // malloc and copy over
    hipMalloc((void**) &d_col_start_collection, threadCount * sizeof(int));
    hipMemcpy(d_col_start_collection, col_start_collection, threadCount * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_col_end_collection, threadCount * sizeof(int));
    hipMemcpy(d_col_end_collection, col_end_collection, threadCount * sizeof(int), hipMemcpyHostToDevice);

    // create a death_toll_collection
    int * d_death_toll_collection;
    int * death_toll_collection;
    hipHostMalloc((void**) &death_toll_collection, sizeof(int) * threadCount * blockCount);
    hipMalloc((void**) &d_death_toll_collection, sizeof(int) * threadCount * blockCount);

#if PRINT_GENERATIONS
    printf("\n=== WORLD 0 ===\n");
    printWorld(startWorld, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
    exportWorld(startWorld, nRows, nCols);
#endif

    // Begin simulating
    int invasionIndex = 0;
    
    // initialise transition world
    int * d_temp_world;
    hipMalloc((void**) &d_temp_world, sizeof(int) * nRows * nCols);
    
    for (int i = 1; i <= nGenerations; i++)
    {
        int * d_invasion = NULL;

#if PRINT_GENERATIONS || EXPORT_GENERATIONS
        int * h_received;
        hipHostMalloc((void**) & h_received, nRows * nCols * sizeof(int));
#endif

        // Check if there's an invasion this particular generation
        if (invasionIndex < nInvasions && i == invasionTimes[invasionIndex])
        {   
            hipMalloc((void**) &d_invasion, sizeof(int) * nRows * nCols);
            hipMemcpy(d_invasion, invasionPlans[invasionIndex], sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
            invasionIndex++;
        }

        // initialise transition world
        int * d_temp_world;
        hipMalloc((void**) &d_temp_world, sizeof(int) * nRows * nCols);
        
        // get new states for each cell
        simulate_per_thread<<<gridDim_, blockDim_>>>(d_world, d_temp_world, d_invasion, d_n_rows, d_n_cols, d_row_start_collection, d_row_end_collection, d_col_start_collection, d_col_end_collection, d_death_toll_collection);
        check_cuda_errors();
        //hipDeviceSynchronize();
        
        // swap the worlds
        hipFree(d_world);
        d_world = d_temp_world;

#if PRINT_GENERATIONS
        hipMemcpy(h_received, d_world, nRows * nCols * sizeof(int), hipMemcpyDeviceToHost);
        printf("\n=== WORLD %d ===\n", i);
        printWorld(h_received, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
        hipMemcpy(h_received, d_world, nRows * nCols * sizeof(int), hipMemcpyDeviceToHost);
        exportWorld(h_received, nRows, nCols);
#endif

#if DEBUG
        printf("One generation passed\n");
#endif

        if (d_invasion != NULL)
        {
            hipFree(d_invasion);
        }

#if PRINT_GENERATIONS || EXPORT_GENERATIONS
        if (h_received != NULL) 
        {
            hipHostFree(h_received);
        }
#endif
        
    }
    
    int deathToll = 0;
    
    hipMemcpy(death_toll_collection, d_death_toll_collection, sizeof(int) * blockCount * threadCount, hipMemcpyDeviceToHost);
    for (int i = 0; i < (blockCount * threadCount); i++) 
    {
        deathToll += death_toll_collection[i];
    }
    
    // clock_gettime(CLOCK_REALTIME, &end);
    // long seconds = end.tv_sec - begin.tv_sec;
    // long nanoseconds = end.tv_nsec - begin.tv_nsec;
    // double elapsed = seconds + nanoseconds * 1e-9;

    // printf("Time taken %.2f\n", elapsed);

    // free arrays on device
    hipFree(d_world); 
    hipFree(d_n_rows);
    hipFree(d_n_cols);
    hipFree(d_col_start_collection);
    hipFree(d_col_end_collection);
    hipFree(d_row_start_collection);
    hipFree(d_row_end_collection);
    hipFree(d_death_toll_collection);

    // free arrays on host
    hipHostFree(col_end_collection);
    hipHostFree(col_start_collection);
    hipHostFree(row_start_collection);
    hipHostFree(row_end_collection);
    hipHostFree(death_toll_collection);
    
    return deathToll;
}
